#include <iostream>
#include <hip/hip_runtime.h>

using namespace std ;

# define DELLEXPORT extern "C" __declspec(dllexport)

__global__ void cudaSquareKernel(float * d_in, float * d_out){
	
	int idx = blockIdx.x * blockIdx.x + threadIdx.x ;
	d_out[idx] = d_in[idx] * d_in[idx] ;
}

DELLEXPORT void cudaSquare(float * h_in, float * h_out, int arr_size){
	
	const long long int ARRAY_BYTES = arr_size * sizeof(float) ;

	float *d_in, *d_out ;

	hipMalloc((void **) &d_in, ARRAY_BYTES) ;
	hipMalloc((void **) &d_out, ARRAY_BYTES) ;

	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice) ;
	
	cudaSquareKernel<<< 1, arr_size >>>(d_in, d_out) ;

	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost) ;

	hipFree(d_in) ;
	hipFree(d_out) ;
}
